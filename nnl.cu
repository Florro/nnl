#include "hip/hip_runtime.h"
/* ToDo
 *
 * 3)  initialization for deep nets
 * 4)  make datalayer construction optional in configurator / generate dim in code and write in config
 * 6)  trainvalidate just one input, and percentage test to train
 * 8)  config for dynamic augmentation
 * 12) save params
 * 13) multithread augmentation?
 * 14) dynamic datastream
 * 17) img to bin
 * 18) background image solution for different colors
 *
 */

/* NOTES
 * cudnn pooling stochastic! -> pooling max, mshadow
 */

#include <time.h>
#include <sys/time.h>
#include "neuralnet/nntrainer.h"
#include "mshadow/tensor.h"


// helper function to messure wall time
double get_wall_time(){
    struct timeval time;
    if (gettimeofday(&time,NULL)){
        //  Handle error
        return 0;
    }
    return (double)time.tv_sec + (double)time.tv_usec * .000001;
}


void read_data_mnist( TensorContainer<cpu, 4, real_t> &xtrain,  TensorContainer<cpu, 4, real_t> &xtest,
		std::vector<int> &ytrain, std::vector<int> &ytest){
	 // settings
	 int insize = 28;
	 srand(0);

	 // data
	 TensorContainer<cpu, 2, real_t> xtrain_, xtest_;
	 utility::LoadMNIST("data/mnist/train-images-idx3-ubyte", "data/mnist/train-labels-idx1-ubyte", ytrain, xtrain_, true);
	 utility::LoadMNIST("data/mnist/t10k-images-idx3-ubyte", "data/mnist/t10k-labels-idx1-ubyte", ytest, xtest_, false);

	 std::cout << std::endl;

	 xtrain.Resize(Shape4(xtrain_.size(0), 1, insize, insize));
	 xtest.Resize(Shape4(xtest_.size(0),  1, insize, insize));
	 xtrain = reshape(xtrain_, xtrain.shape_);
	 xtest = reshape(xtest_, xtest.shape_);
}

// multithreaded run routine
template<typename xpu>
inline int Run(int argc, char *argv[]) {


  TensorContainer<cpu, 4, real_t> xtrain;
  TensorContainer<cpu, 4, real_t> xtest;
  std::vector<int> ytrain, ytest;

  std::string train_path;
  std::string test_path;

  //choose data:
  int data = 1; //0 MNIST, 1 Plankton, 2 retina

  std::vector< std::string > imglst_train;

  //generate nn trainer
  std::string config;
  if(data == 0){
	  read_data_mnist(xtrain, xtest, ytrain, ytest);
	  config = "/home/niklas/CXX/conv_dist_NNL2/testNets/mnist.conf";
  }else if (data == 1){
	  config = "/home/niklas/CXX/conv_dist_NNL2/testNets/plankton.conf";
	  train_path = "/home/niklas/CXX/conv_dist_NNL2/data/plankton/trainnew.lst";
	  test_path = "/home/niklas/CXX/conv_dist_NNL2/data/plankton/testnew.lst";
  }else if (data == 2){
	  //imglst_train = read_data_retina_batch(xtest, ytrain, ytest);
	  //read_data_retina(xtrain, xtest, ytrain, ytest);
	  //config = "/home/niklas/CXX/conv_dist_NNL2/testNets/retina_learning_3class_test.conf";
  }
  nntrainer<xpu>* mynntrainer = new nntrainer<xpu>(argc, argv, config);

  //train routine
  double wall0 = get_wall_time();
  mynntrainer->trainvalidate_batchwise( train_path , test_path, false);
  double wall1 = get_wall_time();

  std::cout << "\nWall Time = " << wall1 - wall0 << std::endl;

  return 0;
}




int main(int argc, char *argv[]) {

  if (argc < 3) {
    printf("Usage: <device> devicelist\n"\
           "\tExample1: ./nnet_ps cpu 1 2 3\n"\
           "\tExample2: ./nnet_ps gpu 0 1\n");
    return 0;
  }
  if (!strcmp(argv[1], "cpu")) {
    Run<mshadow::cpu>(argc, argv);
  } else {
    Run<mshadow::gpu>(argc, argv);
  }


  return 0;

}
